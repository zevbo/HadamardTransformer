#include "hip/hip_runtime.h"
#ifndef __HIPCC__ // If we're not compiling with nvcc or CUDA isn't available
#define __shared__
// #include <thread>
#include <chrono>
#include <cstdio>
#include <ctime>
#define __global__
#define __device__
#define __host__
#define __forceinline__
struct dims {
  int x;
  int y;
  int z;
};
dims threadIdx = {};
dims blockDim = {};
dims blockIdx = {};
dims gridDim = {};
#include "hip/hip_runtime.h"

// #include <immintrin.h>
#else
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <torch/extension.h>
#endif

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <vector>

typedef __half half;
typedef __half2 packed_half;

template <int nFullSize, int nWarpSize, typename ty>
__device__ void load_to_shmem(const ty *x, ty *shmem_x) {
  const ty *block_x = x + nFullSize * blockIdx.x;
  constexpr int32_t nPortion = nFullSize / nWarpSize;
  ty load_registers[nPortion];

#pragma unroll
  for (int32_t i = 0; i < nPortion; i++) {
    load_registers[i] = block_x[threadIdx.x + i * nWarpSize];
  }
#pragma unroll
  for (int32_t i = 0; i < nPortion; i++) {
    shmem_x[threadIdx.x + i * nWarpSize] = load_registers[i];
  }
  __syncthreads();
}

template <int nFullSize, int nWarpSize, typename ty>
__device__ void load_from_shmem(ty *out, const ty *shmem_x) {
  __syncthreads();
  ty *block_out = out + nFullSize * blockIdx.x;
  constexpr int32_t nPortion = nFullSize / nWarpSize;
  ty load_registers[nPortion];
#pragma unroll
  for (int32_t i = 0; i < nPortion; i++) {
    load_registers[i] = shmem_x[threadIdx.x + i * nWarpSize];
  }
#pragma unroll
  for (int32_t i = 0; i < nPortion; i++) {
    block_out[threadIdx.x + i * nWarpSize] = load_registers[i];
  }
}

inline __device__ uint32_t half2_to_uint(packed_half h2_val) {
  return *reinterpret_cast<uint32_t *>(&h2_val);
}

void __device__ tensor_core_hadamard(half *shmem_x) {
  constexpr int side_size = 16;
  int32_t r0 = threadIdx.x / 4;
  int32_t c0 = (threadIdx.x % 4) * 2;
#define is_neg_corn_no_mod(r, c, size) (r >= (size / 2) && c >= (size / 2))
#define is_neg_corn(r, c, size)                                                \
  ((r % size) >= (size / 2) && (c % size) >= (size / 2))
  bool is_neg_0 = is_neg_corn_no_mod(r0, c0, 8) ^ is_neg_corn(r0, c0, 4) ^
                  is_neg_corn(r0, c0, 2);
  float H_0_0 = is_neg_0 ? -1.0f : 1.0f;
  bool is_neg_1 = is_neg_0 ^ is_neg_corn(r0, c0 + 1, 2);
  float H_0_1 = is_neg_1 ? -1.0f : 1.0f;

  packed_half H_0 = __half2(__float2half(H_0_0), __float2half(H_0_1));
  packed_half H_1 = H_0;
  packed_half H_2 = H_0;
  packed_half H_3 = __half2(__float2half(-1 * H_0_0), __float2half(-1 * H_0_1));

  //  constexpr int size = side_size * side_size;
  for (int side = 0; side <= 1; side++) {
    int32_t row_0 = 2 * (threadIdx.x % 4);
    int32_t col_0 = threadIdx.x / 4;
#define get_shmem_x(row, col)                                                  \
  shmem_x[(row) + (col + side * (side_size / 2)) * (side_size)]
    packed_half t_0_1 =
        __half2(get_shmem_x(row_0, col_0), get_shmem_x(row_0 + 1, col_0));
    packed_half t_0_2 = __half2(get_shmem_x(row_0 + side_size / 2, col_0),
                                get_shmem_x(row_0 + side_size / 2 + 1, col_0));

    uint32_t output[2];
    packed_half *packed_half_output = reinterpret_cast<packed_half *>(output);

    asm("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 "
        "{%0, %1}, "
        "{%2, %3, %4, %5}, "
        "{%6, %7}, "
        "{%8, %9};"
        : "=r"(output[0]), "=r"(output[1])
        : "r"(half2_to_uint(H_0)), "r"(half2_to_uint(H_1)),
          "r"(half2_to_uint(H_2)), "r"(half2_to_uint(H_3)),
          "r"(half2_to_uint(t_0_1)), "r"(half2_to_uint(t_0_2)), "r"(0), "r"(0));

    __syncthreads();

    int32_t write_row_0 = threadIdx.x / 4;
    int32_t write_col_0 = (threadIdx.x % 4) * 2;
    get_shmem_x(write_row_0, write_col_0) = __low2half(packed_half_output[0]);
    get_shmem_x(write_row_0, write_col_0 + 1) =
        __high2half(packed_half_output[0]);
    get_shmem_x(write_row_0 + 8, write_col_0) =
        __low2half(packed_half_output[1]);
    get_shmem_x(write_row_0 + 8, write_col_0 + 1) =
        __high2half(packed_half_output[1]);
  }
}

template <int nSize, typename ty> __device__ void simple_hadamard(ty x[nSize]) {
#pragma unroll
  for (int32_t exchange = 1; exchange < nSize; exchange *= 2) {
    int32_t group_size = exchange * 2;
#pragma unroll
    for (int32_t group_i0 = 0; group_i0 < nSize; group_i0 += group_size) {
#pragma unroll
      for (int32_t i = 0; i < exchange; i++) {
        int32_t i0 = group_i0 + i;
        int32_t i1 = i0 + exchange;
        assert(i0 < nSize);
        assert(i1 < nSize);
        ty a = x[i0];
        ty b = x[i1];
        x[i0] = a + b;
        x[i1] = a - b;
      }
    }
  }
}

__global__ void tensor_core_hadamard_256(const half *x, half *out) {

  extern __shared__ float shmem[];
  half *shmem_x = (half *)shmem;
  load_to_shmem<256, 32, half>(x, shmem_x);
  tensor_core_hadamard(shmem_x);
  load_from_shmem<256, 32, half>(out, shmem_x);
}

#define FULL_MASK 0xFFFFFFFF // uint32_t(-1)

template <int nSize, int nWarpSize, typename ty>
__device__ void warp_shuffle_hadamard(ty x[nSize]) {

  int32_t thread_idx = threadIdx.x % nWarpSize;
#pragma unroll
  for (int32_t exchange = 1; exchange < nWarpSize; exchange *= 2) {
    bool is_bottom = (exchange & thread_idx);
#pragma unroll
    for (int32_t i = 0; i < nSize; i++) {
      ty this_val = x[i];
      ty other_x = __shfl_xor_sync(FULL_MASK, this_val, exchange, nWarpSize);
      x[i] = other_x + (is_bottom ? -1 : 1) * this_val;
    }
  }
}

template <int nSize, int nThreads, int nWarpSize, typename ty>
__device__ void interwarp_transpose(ty x[nSize], ty *shmem) {
  constexpr int32_t nWarps = nThreads / nWarpSize;
  int32_t thread_idx = threadIdx.x % nThreads;
  int32_t thread_id = thread_idx % nWarpSize;
  int32_t warp_id = thread_idx / nWarpSize;
  int32_t transposed_thread_id = thread_idx / nWarps;
  int32_t transposed_warp_id = thread_idx % nWarps;
#define index_of(i, thread, warp) (i * nThreads + warp * nWarpSize + thread)

  for (int32_t i = 0; i < nSize; i++) {
    shmem[index_of(i, thread_id, warp_id)] = x[i];
  }
  __syncthreads();
  for (int32_t i = 0; i < nSize; i++) {
    x[i] = shmem[index_of(i, transposed_thread_id, transposed_warp_id)];
  }
}

template <int nSize, int nThreads, int nWarpSize, typename ty>
__device__ void hadamard_transform(ty x[nSize], ty *shmem) {
  constexpr int32_t nWarps = nThreads / nWarpSize;
  simple_hadamard<nSize, ty>(x);
  warp_shuffle_hadamard<nSize, nWarpSize, ty>(x);
  if (nWarps > 1) {
    assert(shmem != nullptr);
    interwarp_transpose<nSize, nThreads, nWarpSize, ty>(x, shmem);
    warp_shuffle_hadamard<nSize, nWarps, ty>(x);
    interwarp_transpose<nSize, nThreads, nWarpSize, ty>(x, shmem);
  }
}

template <int nFullSize, int nWarpSize, typename ty>
__device__ void hadamard_transform_from_shmem(ty *shmem_x) {
  static_assert(nFullSize % nWarpSize == 0,
                "nFullSize must be divisible by nWarpSize");
  constexpr int32_t nSize = nFullSize / nWarpSize;
  ty x[nSize];
  int32_t i0 = threadIdx.x * nSize;
#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i; // ^ threadIdx.x;
    x[j] = shmem_x[i0 + j];
  }

  hadamard_transform<nSize, nWarpSize, nWarpSize, ty>(x, shmem_x);

#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i; // ^ threadIdx.x;
    shmem_x[i0 + j] = x[j];
  }
}

__device__ char float16_to_int4(half val, float scale) {
  float f32 = __half2float(val);
  int scaled = __float2int_rn(f32 * scale);
  return (char)min(max(scaled, -8), 7);
}

__device__ char comb_int4s(char i41, char i42) { return (i41 << 4) + i42; }

template <int nFullSize, int nWarpSize>
__device__ void hadamard_transform_quantize(const half *input_x, char *output) {
  static_assert(nFullSize % nWarpSize == 0,
                "nFullSize must be divisible by nWarpSize");
  constexpr int32_t nSize = nFullSize / nWarpSize;
  static_assert(nSize % 2 == 0,
                "nSize must be a power of 2 (this just checks even though)");
  half x[nSize];
  int32_t thread_idx = threadIdx.x % nWarpSize;
  int32_t i0 = thread_idx * nSize;
#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i; // ^ thread_idx;
    x[j] = input_x[i0 + j];
  }

  hadamard_transform<nSize, nWarpSize, nWarpSize, half>(x, nullptr);

  int32_t i0_out = i0 / 2;

#pragma unroll
  for (int32_t i = 0; i < nSize / 2; i++) {
    int32_t j = i; // ^ thread_idx);
    output[i0_out + j] =
        comb_int4s(float16_to_int4(x[j * 2]), float16_to_int4(x[j * 2 + 1]));
  }
}

template <int nFullSize, int nWarpSize, typename ty>
__global__ void hadamard_transform_from_global(const ty *x, ty *out) {
  extern __shared__ float shmem[];
  ty *shmem_x = (ty *)shmem;

  load_to_shmem<nFullSize, nWarpSize, ty>(x, shmem_x);
  hadamard_transform_from_shmem<nFullSize, nWarpSize, ty>(shmem_x);
  load_from_shmem<nFullSize, nWarpSize, ty>(out, shmem_x);
}

template <int nFullSize> torch::Tensor hadamard_transform_f32(torch::Tensor x) {
  TORCH_CHECK(x.device().type() == torch::kCUDA, "x must be CUDA");
  TORCH_CHECK(x.scalar_type() == torch::kFloat, "Must be f32");
  auto out = torch::empty_like(x);
  int32_t rows = x.size(0);
  printf("Rows, nFullSize: %d, %d\n", rows, nFullSize);
  fflush(stdout);

  auto t1 = std::chrono::high_resolution_clock::now();
  hadamard_transform_from_global<nFullSize, 32, float>
      <<<rows, 32, nFullSize * sizeof(float)>>>(x.data_ptr<float>(),
                                                out.data_ptr<float>());
  hipDeviceSynchronize();
  auto t2 = std::chrono::high_resolution_clock::now();
  auto us =
      std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
  long unsigned int expected_us = ((uint64_t)rows) * ((uint64_t)nFullSize) * 2 *
                                  4 * 1000 * 1000 / (448 * 1024 * 1024);
  expected_us /= 1024;
  float slowdown = (float)us / expected_us;
  printf("Total us: %lu. Ideal: %lu. Slowdown of %.2f\n", us, expected_us,
         slowdown); //,
  //         (float)us / expected_us);
  return out;
}

torch::Tensor hadamard_transform_tensor_core_256(torch::Tensor x) {
  printf("Starting tensor core 256 run\n");
  TORCH_CHECK(x.device().type() == torch::kCUDA, "x must be CUDA");
  TORCH_CHECK(x.scalar_type() == torch::kHalf, "Must be f16");
  auto out = torch::empty_like(x, x.options().dtype(at::kHalf).memory_format(
                                      torch::MemoryFormat::Contiguous));
  printf("Out data ptr: %p\n", out.data_ptr<at::Half>());
  int32_t rows = x.size(0);
  auto t1 = std::chrono::high_resolution_clock::now();
  printf("Getting data pointer\n");
  tensor_core_hadamard_256<<<rows, 32, 256 * sizeof(half)>>>(
      reinterpret_cast<half *>(x.data_ptr<at::Half>()),
      reinterpret_cast<half *>(out.data_ptr<at::Half>()));
  printf("Got data ptr\n");
  printf("Some x stuff: %d, %d. Out stuff: %d, %d\n", x.size(0), x.size(1),
         out.size(0), out.size(1));
  hipDeviceSynchronize();
  auto t2 = std::chrono::high_resolution_clock::now();
  auto us =
      std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
  long unsigned int expected_us = ((uint64_t)rows) * ((uint64_t)256) * 2 * 4 *
                                  1000 * 1000 / (448 * 1024 * 1024);
  expected_us /= 1024;
  float slowdown = (float)us / expected_us;
  printf("Total us: %lu. Ideal: %lu. Slowdown of %.2f\n", us, expected_us,
         slowdown); //,
  return out;
}

torch::Tensor hadamard_transform_f32_512(torch::Tensor x) {
  return hadamard_transform_f32<512>(x);
}

torch::Tensor hadamard_transform_f32_1024(torch::Tensor x) {
  return hadamard_transform_f32<1024>(x);
}
torch::Tensor hadamard_transform_f32_2048(torch::Tensor x) {
  return hadamard_transform_f32<2048>(x);
}

torch::Tensor hadamard_transform_f32_32768(torch::Tensor x) {
  return hadamard_transform_f32<32768>(x);
}

int main() {
  printf("Hello World!\n");
  return 0;
}
