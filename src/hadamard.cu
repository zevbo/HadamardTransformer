#ifndef __CUDACC__ // If we're not compiling with nvcc or CUDA isn't available
#define __shared__
// #include <thread>
#include <ctime>
#define __global__
#define __device__
#define __host__
#define __forceinline__
struct dims {
  int x;
  int y;
  int z;
};
dims threadIdx = {};
dims blockDim = {};
dims blockIdx = {};
dims gridDim = {};
#include "cuda_runtime.h"

// #include <immintrin.h>
#else
#include <hip/hip_runtime.h>
#endif

#include <assert.h>

template <int nSize, typename ty> __device__ void simple_hadamard(ty x[nSize]) {
#pragma unroll
  for (int32_t exchange = 1; exchange < nSize; exchange *= 2) {
    int32_t group_size = exchange << 1;
#pragma unroll
    for (int32_t group = 0; group < nSize; group += group_size) {
      int32_t group_i0 = group * group_size;
#pragma unroll
      for (int32_t i = 0; i < exchange; i++) {
        int32_t i0 = group_i0 + i;
        int32_t i1 = i0 + exchange;
        ty a = x[i0];
        ty b = x[i1];
        x[i0] = a + b;
        x[i1] = a - b;
      }
    }
  }
}

#define FULL_MASK uint32_t(-1)

template <int nSize, int nWarpSize, typename ty>
__device__ void warp_shuffle_hadamard(ty x[nSize]) {

  int32_t thread_idx = threadIdx.x % nWarpSize;
#pragma unroll
  for (int32_t exchange = 1; exchange < nWarpSize; exchange *= 2) {
    int32_t group_size = exchange << 1;
    bool is_bottom = exchange & thread_idx;
#pragma unroll
    for (int32_t i = 0; i < nSize; i++) {
      int32_t this_val = x[i];
      int32_t other_x = __shfl_xor_sync(FULL_MASK, this_val, exchange);
      x[i] = other_x + (is_bottom ? -1 : 1) * x[i];
    }
  }
}

template <int nSize, int nWarpSize, typename ty>
__device__ void hadamard_transform(ty x[nSize]) {
  simple_hadamard<nSize, ty>(x);
  warp_shuffle_hadamard<nSize, ty>(x);
}

template <int nFullSize, int nWarpSize, typename ty>
__device__ void hadamard_transform_from_shmem(ty *shmem_x) {
  if (threadIdx.x >= nWarpSize) {
    // multi-warp not yet supported
    return;
  }
  static_assert(nFullSize % nWarpSize == 0,
                "nFullSize must be divisible by nWarpSize");
  constexpr int32_t nSize = nFullSize / nWarpSize;
  ty x[nSize];
  int32_t i0 = threadIdx.x * nSize;
#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i ^ threadIdx.x;
    x[j] = shmem_x[i0 + j];
  }

  simple_hadamard<nSize, nWarpSize, ty>(x);

#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i ^ threadIdx.x;
    shmem_x[i0 + j] = shmem_x[j];
  }
}

template <int nFullSize, int nWarpSize, typename ty>
__global__ void hadamard_transform_from_global(ty *x) {
  ty *block_x = x + nFullSize * blockIdx.x;
  extern __shared__ float shmem[];
  ty shmem_x[nFullSize] = (ty *)shmem;

  for (int32_t i = threadIdx.x; i < nFullSize; i += blockDim.x) {
    shmem_x[i] = block_x[i];
  }

  hadamard_transform_from_shmem<nFullSize, nWarpSize, ty>(shmem_x);
}
