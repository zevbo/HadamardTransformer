#include "hip/hip_runtime.h"
#ifndef __HIPCC__ // If we're not compiling with nvcc or CUDA isn't available
#define __shared__
// #include <thread>
#include <ctime>
#define __global__
#define __device__
#define __host__
#define __forceinline__
struct dims {
  int x;
  int y;
  int z;
};
dims threadIdx = {};
dims blockDim = {};
dims blockIdx = {};
dims gridDim = {};
#include "hip/hip_runtime.h"

// #include <immintrin.h>
#else
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#endif

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <vector>

typedef __half half;

template <int nSize, typename ty> __device__ void simple_hadamard(ty x[nSize]) {
#pragma unroll
  for (int32_t exchange = 1; exchange < nSize; exchange *= 2) {
    int32_t group_size = exchange * 2;
#pragma unroll
    for (int32_t group_i0 = 0; group_i0 < nSize; group_i0 += group_size) {
#pragma unroll
      for (int32_t i = 0; i < exchange; i++) {
        int32_t i0 = group_i0 + i;
        int32_t i1 = i0 + exchange;
        assert(i0 < nSize);
        assert(i1 < nSize);
        ty a = x[i0];
        ty b = x[i1];
        x[i0] = a + b;
        x[i1] = a - b;
      }
    }
  }
}

#define FULL_MASK 0xFFFFFFFF // uint32_t(-1)

template <int nSize, int nWarpSize, typename ty>
__device__ void warp_shuffle_hadamard(ty x[nSize]) {

  int32_t thread_idx = threadIdx.x % nWarpSize;
#pragma unroll
  for (int32_t exchange = 1; exchange < nWarpSize; exchange *= 2) {
    bool is_bottom = (exchange & thread_idx);
#pragma unroll
    for (int32_t i = 0; i < nSize; i++) {
      ty this_val = x[i];
      ty other_x = __shfl_xor_sync(FULL_MASK, this_val, exchange, nWarpSize);
      x[i] = other_x + (is_bottom ? -1 : 1) * this_val;
    }
  }
}

template <int nSize, int nThreads, int nWarpSize, typename ty>
__device__ void interwarp_transpose(ty x[nSize], ty *shmem) {
  constexpr int32_t nWarps = nThreads / nWarpSize;
  int32_t thread_idx = threadIdx.x % nThreads;
  int32_t thread_id = thread_idx % nWarpSize;
  int32_t warp_id = thread_idx / nWarpSize;
  int32_t transposed_thread_id = thread_idx / nWarps;
  int32_t transposed_warp_id = thread_idx % nWarps;
#define index_of(i, thread, warp) (i * nThreads + warp * nWarpSize + thread)

  for (int32_t i = 0; i < nSize; i++) {
    shmem[index_of(i, thread_id, warp_id)] = x[i];
  }
  __syncthreads();
  for (int32_t i = 0; i < nSize; i++) {
    x[i] = shmem[index_of(i, transposed_thread_id, transposed_warp_id)];
  }
}

template <int nSize, int nThreads, int nWarpSize, typename ty>
__device__ void hadamard_transform(ty x[nSize], ty *shmem) {
  constexpr int32_t nWarps = nThreads / nWarpSize;
  simple_hadamard<nSize, ty>(x);
  warp_shuffle_hadamard<nSize, nWarpSize, ty>(x);
  if (nWarps > 1) {
    assert(shmem != nullptr);
    interwarp_transpose<nSize, nThreads, nWarpSize, ty>(x, shmem);
    warp_shuffle_hadamard<nSize, nWarps, ty>(x);
    interwarp_transpose<nSize, nThreads, nWarpSize, ty>(x, shmem);
  }
}

template <int nFullSize, int nWarpSize, typename ty>
__device__ void hadamard_transform_from_shmem(ty *shmem_x) {
  static_assert(nFullSize % nWarpSize == 0,
                "nFullSize must be divisible by nWarpSize");
  constexpr int32_t nSize = nFullSize / nWarpSize;
  ty x[nSize];
  int32_t i0 = threadIdx.x * nSize;
#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i; // ^ threadIdx.x;
    x[j] = shmem_x[i0 + j];
  }

  hadamard_transform<nSize, nWarpSize, nWarpSize, ty>(x, shmem_x);

#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i; // ^ threadIdx.x;
    shmem_x[i0 + j] = x[j];
  }
}

__device__ char float16_to_int4(half val, float scale) {
  float f32 = __half2float(val);
  int scaled = __float2int_rn(f32 * scale);
  return (char)min(max(scaled, -8), 7);
}

__device__ char comb_int4s(char i41, char i42) { return (i41 << 4) + i42; }

template <int nFullSize, int nWarpSize>
__device__ void hadamard_transform_quantize(const half *input_x, char *output) {
  static_assert(nFullSize % nWarpSize == 0,
                "nFullSize must be divisible by nWarpSize");
  constexpr int32_t nSize = nFullSize / nWarpSize;
  static_assert(nSize % 2 == 0,
                "nSize must be a power of 2 (this just checks even though)");
  half x[nSize];
  int32_t thread_idx = threadIdx.x % nWarpSize;
  int32_t i0 = thread_idx * nSize;
#pragma unroll
  for (int32_t i = 0; i < nSize; i++) {
    int32_t j = i; // ^ thread_idx;
    x[j] = input_x[i0 + j];
  }

  hadamard_transform<nSize, nWarpSize, nWarpSize, half>(x, nullptr);

  int32_t i0_out = i0 / 2;

#pragma unroll
  for (int32_t i = 0; i < nSize / 2; i++) {
    int32_t j = i; // ^ thread_idx);
    output[i0_out + j] =
        comb_int4s(float16_to_int4(x[j * 2]), float16_to_int4(x[j * 2 + 1]));
  }
}

template <int nFullSize, int nWarpSize, typename ty>
__global__ void hadamard_transform_from_global(const ty *x, ty *out) {
  if (blockIdx.x > 0) {
    return;
  }
  const ty *block_x = x + nFullSize * blockIdx.x;
  ty *block_out = out + nFullSize * blockIdx.x;
  extern __shared__ float shmem[];
  ty *shmem_x = (ty *)shmem;

  for (int32_t i = threadIdx.x; i < nFullSize; i += blockDim.x) {
    if (blockIdx.x == 1) {
      // assert(block_x[i] == 0);
    }
    shmem_x[i] = block_x[i];
  }

  hadamard_transform_from_shmem<nFullSize, nWarpSize, ty>(shmem_x);

  for (int32_t i = threadIdx.x; i < nFullSize; i += blockDim.x) {
    block_out[i] = shmem_x[i];
  }
}

torch::Tensor hadamard_transform_f32_1024(torch::Tensor x, int rows) {
  TORCH_CHECK(x.device().type() == torch::kCUDA, "x must be CUDA");
  TORCH_CHECK(x.scalar_type() == torch::kFloat, "Must be f32");
  auto out = torch::empty_like(x);
  int32_t rows_ = x.size(0);
  int32_t cols = x.size(1);
  printf("Rows, cols: %d, %d\n", rows_, cols);
  fflush(stdout);
  hadamard_transform_from_global<1024, 32, float>
      <<<rows, 32, 1024 * 48>>>(x.data_ptr<float>(), out.data_ptr<float>());
  return out;
}

int main() {
  printf("Hello World!\n");
  return 0;
}
